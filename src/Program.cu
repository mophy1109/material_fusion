#include "hip/hip_runtime.h"
/*
 * @Author: USTB.mophy1109
 * @Date: 2018-05-02 11:30:14
 * @Last Modified by: USTB.mophy1109
 * @Last Modified time: 2018-05-29 11:23:20
 */

#include <opencv2/core/cuda.hpp>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;
using namespace cv::cuda;

//define the image block size to calculate SF,
#define SUB_BLOCK_WIDTH 4
#define SUB_BLOCK_HEIGHT 4

__global__ void calSF(
    const PtrStepSz<uchar> img1,
    const PtrStepSz<uchar> img2,
    PtrStepSz<uchar> result)
    {
    //calculate Spatial Frequency of images and return the weight-matrix

    __shared__ int RF1;//block shared memory for save RF1 in a block
    __shared__ int CF1;//block shared memory for save CF1 in a block
    __shared__ int RF2;//block shared memory for save RF2 in a block
    __shared__ int CF2;//block shared memory for save CF2 in a block

    RF1 = CF1 = RF2 = CF2 = 0;
    __syncthreads();

    /*
      Cause img1 and img2 have the same block size,
      only need to calculate CF^2 and RF^2 to compare.
      so RF & CF here are actually RF^2 and CF^2
    */

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < img1.cols && y < img1.rows)
    {   
        //using aromic ops to avoid conflict reading-modification-writing ops
        if (y > 0){
            atomicAdd(&CF1, (img1(y,x) - img1(y-1,x))*(img1(y,x) - img1(y-1,x)));
            atomicAdd(&CF2, (img2(y,x) - img2(y-1,x))*(img2(y,x) - img2(y-1,x)));
        }
        if (x > 0){
            atomicAdd(&RF1, (img1(y,x) - img1(y,x-1))*(img1(y,x) - img1(y,x-1)));
            atomicAdd(&RF2, (img2(y,x) - img2(y,x-1))*(img2(y,x) - img2(y,x-1)));
        }
        __syncthreads();

        //generate weight_matrix, no need to use atomic ops cause thread in the same block get the same result
        if (RF1 + CF1 > RF2 + CF2){
            result(y/SUB_BLOCK_HEIGHT, x/SUB_BLOCK_WIDTH) = 1;
        }else{
            result(y/SUB_BLOCK_HEIGHT, x/SUB_BLOCK_WIDTH) = 0;
        }
    }
}

void calculateSF_caller(const PtrStepSz<uchar> &img1, const PtrStepSz<uchar> &img2, PtrStepSz<uchar> result, hipStream_t stream)
{
    // define the block size
    dim3 block(SUB_BLOCK_WIDTH,SUB_BLOCK_HEIGHT);

    // define the grid size, which is dynamically calculated with the size of img
    dim3 grid((img1.cols + block.x - 1)/block.x,(img1.rows + block.y - 1)/block.y);

    calSF<<<grid, block>>>(img1, img2, result);//get RF&CF of img1 and img2
    hipDeviceSynchronize();
}