#include "hip/hip_runtime.h"
/*
 * @Author: USTB.mophy1109
 * @Date: 2018-05-02 11:30:14
 * @Last Modified by: USTB.mophy1109
 * @Last Modified time: 2018-05-18 10:37:40 
 */

#include "program.h"
#include <opencv2/core/cuda.hpp>
#include <hip/hip_runtime.h>
#include <CUtil.h>

//default block size, same to block size in image:4*4
#define block_width 4
#define block_height 4


using namespace std;
using namespace cv;
using namespace cv::cuda;


//define the image block size to calculate SF,
#define SUB_BLOCK_WIDTH 4
#define SUB_BLOCK_HEIGHT 4

__global__ void calSF(
    const PtrStepSz<uchar> img1,
    const PtrStepSz<uchar> img2,
    PtrStepSz<uchar> result)
    {
    //calculate Spatial Frequency of images and return the 

    __shared__ int RF1;//block shared memory for save RF1 in a block
    __shared__ int CF1;//block shared memory for save CF1 in a block
    __shared__ int RF2;//block shared memory for save RF2 in a block
    __shared__ int CF2;//block shared memory for save CF2 in a block

    RF1 = CF1 = RF2 = CF2 = 0;
    __syncthreads();

    /*
      Because img1 and img2 have the same block size,
      we only need to calculate CF^2 and RF^2 to compare.
      so RF & CF here are actually RF^2 and CF^2
    */

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < img1.cols && y < img1.rows)
    {   
        if (y > 0){
            atomicAdd(&CF1, (img1(y,x) - img1(y-1,x))*(img1(y,x) - img1(y-1,x)));
            atomicAdd(&CF2, (img2(y,x) - img2(y-1,x))*(img2(y,x) - img2(y-1,x)));
        }
        if (x > 0){
            atomicAdd(&RF1, (img1(y,x) - img1(y,x-1))*(img1(y,x) - img1(y,x-1)));
            atomicAdd(&RF2, (img2(y,x) - img2(y,x-1))*(img2(y,x) - img2(y,x-1)));
        }
        __syncthreads();

        if (RF1 + CF1 > RF2 + CF2){
            result(y/SUB_BLOCK_HEIGHT, x/SUB_BLOCK_WIDTH) = 1;
        }else{
            result(y/SUB_BLOCK_HEIGHT, x/SUB_BLOCK_WIDTH) = 0;
        }
    }
}

void calculateSF_caller(const PtrStepSz<uchar> &img1, const PtrStepSz<uchar> &img2, PtrStepSz<uchar> result, hipStream_t stream)
{
    dim3 block(SUB_BLOCK_WIDTH,SUB_BLOCK_HEIGHT);
    dim3 grid((img1.cols + block.x - 1)/block.x,(img1.rows + block.y - 1)/block.y);
    calSF<<<grid, block>>>(img1, img2, result);//get RF&CF of img1 and img2
    hipDeviceSynchronize();
}